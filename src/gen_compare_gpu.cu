#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <vector>

#include "gen_compare_gpu.h"

__global__ void findMatchesKernel(int* genes1, int* genes2, int* result, int min, int maxcg1, int maxcg2) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x; // thread id
    // gen conts
    int cg1 = tid / maxcg2;
    int cg2 = tid % maxcg2; // así conseguimos que cada thread tenga un par de contadores únicos

    // no hacemos solapamientos en caso de que N_thread > N_blocks
    if (cg1 >= maxcg1 || cg2 >= maxcg2) {
        return;
    }

    // básicamente es que, una vez se sale del bucle, tenemos una coincidencia en el rango [cgX, icgX]
    int icg1 = cg1;
    int icg2 = cg2;
    while (icg1 < maxcg1 && icg2 < maxcg2 && genes1[icg1] == genes2[icg2]) {
        ++icg1;
        ++icg2;
    }

    int gl = icg1 - cg1; // gen len
    if (gl >= min) {
        int index = atomicAdd(result, 3); // no WAW
        result[index + 1] = cg1;
        result[index + 2] = cg2;
        result[index + 3] = gl;
    }
}

std::vector<int> findMatchesGPU(GeneSequence &genes1, GeneSequence &genes2, int min) {
    int maxcg1 = genes1.size() - min; // si str de len 10 y min 4, entonces el índice máx es 10 - 4 = 6 para no hacer un OOB
    int maxcg2 = genes2.size() - min;

    // cast a * para la GPU
    int* d_genes1;
    int* d_genes2;
    hipMalloc(&d_genes1, genes1.size() * sizeof(int));
    hipMalloc(&d_genes2, genes2.size() * sizeof(int));
    hipMemcpy(d_genes1, genes1.data(), genes1.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_genes2, genes2.data(), genes2.size() * sizeof(int), hipMemcpyHostToDevice);

    // para el vector [[g1, g2, len], [,,,], ...]
    int* d_result;
    hipMalloc(&d_result, 3 * maxcg1 * maxcg2 * sizeof(int));
    hipMemset(d_result, 0, 3 * maxcg1 * maxcg2 * sizeof(int));

    // GTX 3070 TI
    int blockSize = 256;
    int gridSize = (maxcg1 * maxcg2 + blockSize - 1) / blockSize;
    findMatchesKernel<<<gridSize, blockSize>>>(d_genes1, d_genes2, d_result, min, maxcg1, maxcg2);

    // ver el número de resultados
    int resultSize;
    hipMemcpy(&resultSize, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // un copy de toda la vida
    std::vector<int> result(resultSize);
    hipMemcpy(result.data(), d_result + 1, resultSize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_genes1);
    hipFree(d_genes2);
    hipFree(d_result);

    return result;
}
